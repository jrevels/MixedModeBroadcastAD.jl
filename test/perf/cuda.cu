#include <cstdlib>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      std::cerr << "CUDA Error: " << hipGetErrorString(stat) << " " << file << " " << line << std::endl;
   }
}


//
// Fused
//

__forceinline__ __device__ float sigmoidf(float in) {
     return 1.f / (1.f + expf(-in));  
}

__global__ void fused_lstm_update_c(int numElements, float* out,
                                    const float* c,
                                    const float* Wx_f, const float* Wx_i, const float* Wx_c,
                                    const float* Rh_f, const float* Rh_i, const float* Rh_c,
                                    const float* b_f,  const float* b_i,  const float* b_c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numElements) return;

    out[index] = sigmoidf(Wx_f[index] + Rh_f[index] + b_f[index]) * c[index] +
                 sigmoidf(Wx_i[index] + Rh_i[index] + b_i[index]) *
                 tanh(Wx_c[index] + Rh_c[index] + b_c[index]);
}


//
// Unfused
//

__global__ void pw_tanh(float *y, const float *a, int numElements) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < numElements) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, const float *a, int numElements) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < numElements) y[i] = sigmoidf(a[i]);
}

__global__ void pw_vecAdd(float *y, const float *a,  const float *b, int numElements) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < numElements) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, const float *a,  const float *b, int numElements) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < numElements) y[i] = a[i] * b[i];
}

void unfused_lstm_update_c(int numElements, float* out,
                           const float* c,
                           const float* Wx_f, const float* Wx_i, const float* Wx_c,
                           const float* Rh_f, const float* Rh_i, const float* Rh_c,
                           const float* b_f,  const float* b_i,  const float* b_c) {

    dim3 blockDim;
    dim3 gridDim;

    blockDim.x = 256;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

    float *tmp1, *tmp2;
    cudaErrCheck(hipMalloc((void**)&tmp1, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&tmp2, numElements * sizeof(float)));

    // sigmoid(Wx_f + Rh_f + b_f) * c
    pw_vecAdd<<<gridDim, blockDim>>>(tmp1, Wx_f, Rh_f, numElements);
    pw_vecAdd<<<gridDim, blockDim>>>(tmp1, tmp1, b_f, numElements);
    pw_sigmoid<<<gridDim, blockDim>>>(tmp1, tmp1, numElements);
    pw_vecMul<<<gridDim, blockDim>>>(tmp1, tmp1, c, numElements);

    // sigmoid(Wx_i + Rh_i + b_i)
    pw_vecAdd<<<gridDim, blockDim>>>(tmp2, Wx_i, Rh_i, numElements);
    pw_vecAdd<<<gridDim, blockDim>>>(tmp2, tmp2, b_i, numElements);
    pw_sigmoid<<<gridDim, blockDim>>>(tmp2, tmp2, numElements);

    // tanh(Wx_c + Rh_c + b_c)
    pw_vecAdd<<<gridDim, blockDim>>>(out, Wx_c, Rh_c, numElements);
    pw_vecAdd<<<gridDim, blockDim>>>(out, out, b_c, numElements);
    pw_tanh<<<gridDim, blockDim>>>(out, out, numElements);

    // sigmoid(...) * tanh(...)
    pw_vecMul<<<gridDim, blockDim>>>(out, out, tmp2, numElements);

    // sigmoid(...) + sigmoid(...) * tanh(...)
    pw_vecAdd<<<gridDim, blockDim>>>(out, out, tmp1, numElements);

    cudaErrCheck(hipFree(tmp1));
    cudaErrCheck(hipFree(tmp2));
}


//
// Entry-points
//

extern "C" void execute(int numElements, int fused, float* out,
                        const float* c,
                        const float* Wx_f, const float* Wx_i, const float* Wx_c,
                        const float* Rh_f, const float* Rh_i, const float* Rh_c,
                        const float* b_f,  const float* b_i,  const float* b_c) {
    if (fused) {
        dim3 blockDim;
        dim3 gridDim;

        blockDim.x = 256;
        gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

        fused_lstm_update_c<<<gridDim, blockDim>>>(numElements, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);
    } else {
        unfused_lstm_update_c(numElements, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);
    }
}

extern "C" float benchmark(int n, int fused) {
    int numElements = n*n;

    float *out, *c, *Wx_f, *Wx_i, *Wx_c, *Rh_f, *Rh_i, *Rh_c, *b_f, *b_i, *b_c;
    cudaErrCheck(hipMalloc((void**)&c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&out, numElements * sizeof(float)));

    dim3 blockDim;
    dim3 gridDim;

    blockDim.x = 256;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

    auto start = std::chrono::system_clock::now();
    execute(numElements, fused, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();

    std::chrono::duration<float> elapsed = end - start;
    return elapsed.count();
}


//
// Main
//

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " N" << std::endl;
        return EXIT_FAILURE;
    }
    int n = atoi(argv[1]);
    std::cout << "Fused: " << benchmark(n, 1) << std::endl;
    std::cout << "Unfused: " << benchmark(n, 0) << std::endl;
    return EXIT_SUCCESS;
}
