#include <hip/hip_runtime.h>
#include <iostream>

#define cudaErrCheck(stat)                                                     \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    std::cerr << "CUDA Error: " << hipGetErrorString(stat) << " " << file
              << " " << line << std::endl;
  }
}

//
// Fused
//

__forceinline__ __device__ float sigmoidf(float in) {
  return 1.f / (1.f + expf(-in));
}

__global__ void lstm_update_c_kernel(int numElements, float *out, const float *c,
                                    const float *Wx_f, const float *Wx_i,
                                    const float *Wx_c, const float *Rh_f,
                                    const float *Rh_i, const float *Rh_c,
                                    const float *b_f, const float *b_i,
                                    const float *b_c) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= numElements)
    return;

  out[index] = sigmoidf(Wx_f[index] + Rh_f[index] + b_f[index]) * c[index] +
               sigmoidf(Wx_i[index] + Rh_i[index] + b_i[index]) *
                   tanh(Wx_c[index] + Rh_c[index] + b_c[index]);
}

extern "C" void lstm_update_c(int numElements, float *out, const float *c, const float *Wx_f,
                        const float *Wx_i, const float *Wx_c, const float *Rh_f,
                        const float *Rh_i, const float *Rh_c, const float *b_f,
                        const float *b_i, const float *b_c) {
    dim3 blockDim;
    dim3 gridDim;

    blockDim.x = 256;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

    lstm_update_c_kernel<<<gridDim, blockDim>>>(
        numElements, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);

}


//
// Unfused
//

__global__ void pw_tanh(float *y, const float *a, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, const float *a, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = sigmoidf(a[i]);
}

__global__ void pw_vecAdd(float *y, const float *a, const float *b,
                          int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, const float *a, const float *b,
                          int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = a[i] * b[i];
}

extern "C" void unfused_lstm_update_c(int numElements, float *out, float *tmp1,
                                      float *tmp2, const float *c, const float *Wx_f,
                                      const float *Wx_i, const float *Wx_c,
                                      const float *Rh_f, const float *Rh_i,
                                      const float *Rh_c, const float *b_f,
                                      const float *b_i, const float *b_c) {
  dim3 blockDim;
  dim3 gridDim;

  blockDim.x = 256;
  gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

  // sigmoid(Wx_f + Rh_f + b_f) * c
  pw_vecAdd<<<gridDim, blockDim>>>(tmp1, Wx_f, Rh_f, numElements);
  pw_vecAdd<<<gridDim, blockDim>>>(tmp1, tmp1, b_f, numElements);
  pw_sigmoid<<<gridDim, blockDim>>>(tmp1, tmp1, numElements);
  pw_vecMul<<<gridDim, blockDim>>>(tmp1, tmp1, c, numElements);

  // sigmoid(Wx_i + Rh_i + b_i)
  pw_vecAdd<<<gridDim, blockDim>>>(tmp2, Wx_i, Rh_i, numElements);
  pw_vecAdd<<<gridDim, blockDim>>>(tmp2, tmp2, b_i, numElements);
  pw_sigmoid<<<gridDim, blockDim>>>(tmp2, tmp2, numElements);

  // tanh(Wx_c + Rh_c + b_c)
  pw_vecAdd<<<gridDim, blockDim>>>(out, Wx_c, Rh_c, numElements);
  pw_vecAdd<<<gridDim, blockDim>>>(out, out, b_c, numElements);
  pw_tanh<<<gridDim, blockDim>>>(out, out, numElements);

  // sigmoid(...) * tanh(...)
  pw_vecMul<<<gridDim, blockDim>>>(out, out, tmp2, numElements);

  // sigmoid(...) + sigmoid(...) * tanh(...)
  pw_vecAdd<<<gridDim, blockDim>>>(out, out, tmp1, numElements);
}
