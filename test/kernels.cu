#include <hip/hip_runtime.h>
#include <iostream>

#define cudaErrCheck(stat)                                                     \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    std::cerr << "CUDA Error: " << hipGetErrorString(stat) << " " << file
              << " " << line << std::endl;
  }
}

//
// Fused
//

__forceinline__ __device__ float sigmoidf(float in) {
  return 1.f / (1.f + expf(-in));
}

__global__ void lstm_update_c_kernel(int numElements, float *out,
                                     const float *c, const float *Wx_f,
                                     const float *Wx_i, const float *Wx_c,
                                     const float *Rh_f, const float *Rh_i,
                                     const float *Rh_c, const float *b_f,
                                     const float *b_i, const float *b_c) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= numElements)
    return;

  out[index] = sigmoidf(Wx_f[index] + Rh_f[index] + b_f[index]) * c[index] +
               sigmoidf(Wx_i[index] + Rh_i[index] + b_i[index]) *
                   tanh(Wx_c[index] + Rh_c[index] + b_c[index]);
}

extern "C" void lstm_update_c(int numElements, float *out, const float *c,
                              const float *Wx_f, const float *Wx_i,
                              const float *Wx_c, const float *Rh_f,
                              const float *Rh_i, const float *Rh_c,
                              const float *b_f, const float *b_i,
                              const float *b_c) {
  dim3 blockDim;
  dim3 gridDim;

  blockDim.x = 256;
  gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

  lstm_update_c_kernel<<<gridDim, blockDim>>>(
      numElements, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);
}

//
// Unfused
//

__global__ void pw_tanh(float *y, const float *a, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, const float *a, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = sigmoidf(a[i]);
}

__global__ void pw_vecAdd2(float *y, const float *a, const float *b,
                           int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = a[i] + b[i];
}

__global__ void pw_vecAdd3(float *y, const float *a, const float *b,
                           const float *c, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = a[i] + b[i] + c[i];
}

__global__ void pw_vecMul(float *y, const float *a, const float *b,
                          int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements)
    y[i] = a[i] * b[i];
}

extern "C" void
unfused_lstm_update_c(int numElements, float *out, float *tmp1, float *tmp2,
                      float *tmp3, float *tmp4, float *tmp5, float *tmp6,
                      float *tmp7, float *tmp8, const float *c,
                      const float *Wx_f, const float *Wx_i, const float *Wx_c,
                      const float *Rh_f, const float *Rh_i, const float *Rh_c,
                      const float *b_f, const float *b_i, const float *b_c) {
  dim3 blockDim;
  dim3 gridDim;

  blockDim.x = 256;
  gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

  // sigmoid(Wx_f + Rh_f + b_f) * c
  pw_vecAdd3<<<gridDim, blockDim>>>(tmp1, Wx_f, Rh_f, b_f, numElements);
  pw_sigmoid<<<gridDim, blockDim>>>(tmp2, tmp1, numElements);
  pw_vecMul<<<gridDim, blockDim>>>(tmp3, tmp2, c, numElements);

  // sigmoid(Wx_i + Rh_i + b_i)
  pw_vecAdd3<<<gridDim, blockDim>>>(tmp4, Wx_i, Rh_i, b_i, numElements);
  pw_sigmoid<<<gridDim, blockDim>>>(tmp5, tmp4, numElements);

  // tanh(Wx_c + Rh_c + b_c)
  pw_vecAdd3<<<gridDim, blockDim>>>(tmp6, Wx_c, Rh_c, b_c, numElements);
  pw_tanh<<<gridDim, blockDim>>>(tmp7, tmp6, numElements);

  // sigmoid(...) * tanh(...)
  pw_vecMul<<<gridDim, blockDim>>>(tmp8, tmp5, tmp7, numElements);

  // sigmoid(...) + sigmoid(...) * tanh(...)
  pw_vecAdd2<<<gridDim, blockDim>>>(out, tmp3, tmp8, numElements);
}