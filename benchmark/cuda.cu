#include <cstdlib>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>

#define N 2048

__forceinline__ __device__ float sigmoidf(float in) {
     return 1.f / (1.f + expf(-in));  
}

__global__ void cuda_lstm_update_c(int n, float* out,
                                   const float* c,
                                   const float* Wx_f, const float* Wx_i, const float* Wx_c,
                                   const float* Rh_f, const float* Rh_i, const float* Rh_c,
                                   const float* b_f,  const float* b_i,  const float* b_c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    out[index] = sigmoidf(Wx_f[index] + Rh_f[index] + b_f[index]) * c[index] +
                 sigmoidf(Wx_i[index] + Rh_i[index] + b_i[index]) *
                 tanh(Wx_c[index] + Rh_c[index] + b_c[index]);
}

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      std::cerr << "CUDA Error: " << hipGetErrorString(stat) << " " << file << " " << line << std::endl;
   }
}

extern "C" float benchmark(int n) {
    int numElements = n*n;

    float *out, *c, *Wx_f, *Wx_i, *Wx_c, *Rh_f, *Rh_i, *Rh_c, *b_f, *b_i, *b_c;
    cudaErrCheck(hipMalloc((void**)&c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_f, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_i, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Wx_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&Rh_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&b_c, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&out, numElements * sizeof(float)));

    dim3 blockDim;
    dim3 gridDim;

    blockDim.x = 256;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

    auto start = std::chrono::system_clock::now();
    cuda_lstm_update_c<<<gridDim, blockDim>>>(numElements, out, c, Wx_f, Wx_i, Wx_c, Rh_f, Rh_i, Rh_c, b_f, b_i, b_c);
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();

    std::chrono::duration<float> elapsed = end - start;
    return elapsed.count();
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " N" << std::endl;
        return EXIT_FAILURE;
    }
    int n = atoi(argv[1]);
    std::cout << benchmark(n) << std::endl;
    return EXIT_SUCCESS;
}
